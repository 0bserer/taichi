#include <iostream>
#include <cstdlib>
#include <cstdio>
#include <hip/hip_runtime.h>
#include <sys/time.h>

double get_time() {
  struct timeval tv;
  gettimeofday(&tv, nullptr);
  return tv.tv_sec + 1e-6 * tv.tv_usec;
}

constexpr int m = 256;
constexpr int block_size = 128;

struct Node {
  int lock;
  int sum;
};

__global__ void inc(Node *nodes) {
  unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

  int b = i * 3 % m;

  int warp_id = threadIdx.x % 32;
  int done = 0;
  auto mask = __activemask();
  // printf("mask %d\n", mask);
  while (!__all_sync(mask, done)) {
    for (int k = 0; k < 32; k++) {
      if (k == warp_id && !done) {
        int &lock = nodes[b].lock;
        if (atomicCAS(&lock, 0, 1) == 0) {
          nodes[b].sum += 1;
          done = true;
          atomicExch(&lock, 0);
        }
      }
    }
  }
}

int main() {
  Node *a;

  hipMallocManaged(&a, m * sizeof(Node));

  for (int i = 0; i < 20; i++) {
    hipDeviceSynchronize();
    auto t = get_time();
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    hipDeviceSynchronize();
    inc<<<m, block_size>>>((Node *)a);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    std::cout << "device  " << milliseconds << std::endl;
    int sum = 0;
    for (int j = 0; j < m; j++) {
      sum += a[j].sum;
    }
    printf("sum %d\n", sum);
  }
  std::cout << std::endl;
}
