#include "hip/hip_runtime.h"
#include <cstdio>
#include <cassert>

__global__ void init_random_numbers(unsigned int seed) {
  printf("seed = %d\n", seed);
  atomicAdd(int *(12312433432), 123);
  assert(seed != 0);
}

int main() {
  init_random_numbers<<<1024, 1024>>>(1);
  return 0;
}
